#include "hip/hip_runtime.h"
#include "KNN_CUDA.cuh"

__constant__ float constant_query[100 * DATA_SIZE];

__global__ void min_distance_coalesced_sh_query_constant(float* dataset, float* results, int dataset_size, int data_size, int query_size) {

	__shared__ float sh_dataset[DATA_SIZE];
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0;
	float v = 0;

	sh_dataset[threadIdx.x] = dataset[row];
	__syncthreads();

	if (threadIdx.x < query_size) {
		for (int i = 0; i < DATA_SIZE; ++i) {
			v = sh_dataset[i] - constant_query[i + threadIdx.x * DATA_SIZE];
			d += v * v;
		}

		results[  blockIdx.x + threadIdx.x * dataset_size] = d;
		__syncthreads();

	}
}

void min_dinstance_coalesced_shDS_constQ_cuda(float* dataset, vector<float> query, thrust::device_vector<float> result, int* result_index, int dataset_size, int data_size, int query_size, int block_size) {

	cout << "============= MIN DISTANCE COALESCED SH DATASET CONST QUERY ============" << endl << endl;

	dim3 dim_grid_min_dist = dataset_size;
	dim3 dim_block_min_dist = 128;

	double startGPU;
	float* constant_query_ptr;
	int phase = ceil(query_size / 100);
	int query_size_tmp = 0;
	float* result_ptr;
	int b;
	startGPU = omp_get_wtime();

	for (int i = 0; i < phase; i++) {
		if (query_size_tmp + 100 <= query_size)
			query_size_tmp += 100;
		else {
			query_size_tmp += query_size % 100;
			b = query_size_tmp;
			dim_grid_min_dist = ceil((float)dataset_size / b);
			dim_block_min_dist = b;
		}

		result_ptr = thrust::raw_pointer_cast(&result[i * 100 * dataset_size]);
		constant_query_ptr = &query[i * 100 * data_size];
		hipMemcpyToSymbol(HIP_SYMBOL(constant_query), constant_query_ptr, 100 * data_size * sizeof(float));
		min_distance_coalesced_sh_query_constant << < dim_grid_min_dist, dim_block_min_dist >> > (dataset, result_ptr, dataset_size, data_size, query_size_tmp);
		hipDeviceSynchronize();
	}
	result_ptr = thrust::raw_pointer_cast(&result[0]);
	insertion_sort_cuda(result_ptr, result_index, dataset_size, query_size);
	hipDeviceSynchronize();
	printf_s("TIME: %.16g\n\n", omp_get_wtime() - startGPU);
}